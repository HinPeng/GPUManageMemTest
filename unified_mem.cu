#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <math.h>

#include <hip/hip_runtime_api.h>
 
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y, float *u, float *z)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i] + u[i] + z[i];
    // y[i] = x[i] + y[i];
}
 
int main(void)
{
  size_t N = static_cast<long long>(1) << 26;
  size_t mem_size = N * sizeof(float);

  #ifdef _MANAGEMEMORY
  float *x, *y, *u, *z;
  // float *x, *y;
  
  // Allocate Unified Memory -- accessible from CPU or GPU
  checkCudaErrors(hipMallocManaged(&x, mem_size));
  checkCudaErrors(hipMallocManaged(&y, mem_size));
  checkCudaErrors(hipMallocManaged(&u, mem_size));
  checkCudaErrors(hipMallocManaged(&z, mem_size));
  #else
  float *h_x, *h_y, *h_u, *h_z;
  // float *h_x, *h_y;
  h_x = (float *)malloc(mem_size);
  h_y = (float *)malloc(mem_size);
  h_u = (float *)malloc(mem_size);
  h_z = (float *)malloc(mem_size);

  float *d_x, *d_y, *d_u, *d_z;
  // float *d_x, *d_y;
  checkCudaErrors(hipMalloc((void**) &d_x, mem_size));
  checkCudaErrors(hipMalloc((void**) &d_y, mem_size));
  checkCudaErrors(hipMalloc((void**) &d_u, mem_size));
  checkCudaErrors(hipMalloc((void**) &d_z, mem_size));
  #endif
 
  // initialize x and y arrays on the host
  for (long long i = 0; i < N; i++) {
    #ifdef _MANAGEMEMORY
    x[i] = 1.0f;
    y[i] = 2.0f;
    u[i] = 3.0f;
    z[i] = 4.0f;
    #else
    h_x[i] = 1.0f;
    h_y[i] = 2.0f;
    h_u[i] = 3.0f;
    h_z[i] = 4.0f;
    #endif
  }
 
  int device = -1;
  hipGetDevice(&device);
  #ifdef _USEPREFETCH
  checkCudaErrors(hipMemPrefetchAsync(x, mem_size, device, NULL));
  checkCudaErrors(hipMemPrefetchAsync(y, mem_size, device, NULL));
  checkCudaErrors(hipMemPrefetchAsync(u, mem_size, device, NULL));
  checkCudaErrors(hipMemPrefetchAsync(z, mem_size, device, NULL));
  #else
  #ifndef _MANAGEMEMORY
  checkCudaErrors(hipMemcpy(d_x, h_x, mem_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, mem_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_u, h_u, mem_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_z, h_z, mem_size, hipMemcpyHostToDevice));
  #endif
  #endif

  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipEventRecord(start, NULL));
  #ifdef _MANAGEMEMORY
  add<<<numBlocks, blockSize>>>(N, x, y, u, z);
  // add<<<numBlocks, blockSize>>>(N, x, y);
  #else
  add<<<numBlocks, blockSize>>>(N, d_x, d_y, d_u, d_z);
  // add<<<numBlocks, blockSize>>>(N, d_x, d_y);
  #endif

  checkCudaErrors(hipEventRecord(stop, NULL));
  // Wait for GPU to finish before accessing on host
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  printf("Time = %.3f msec\n", msecTotal);
 
  #ifndef _MANAGEMEMORY
  checkCudaErrors(hipMemcpy(h_y, d_y, mem_size, hipMemcpyDeviceToHost));
  #endif
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    #ifdef _MANAGEMEMORY
    maxError = fmax(maxError, fabs(y[i]-10.0f));
    #else
    maxError = fmax(maxError, fabs(h_y[i]-10.0f));
    #endif
  std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  #ifdef _MANAGEMEMORY
  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(y));
  checkCudaErrors(hipFree(u));
  checkCudaErrors(hipFree(z));
  #else
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_u));
  checkCudaErrors(hipFree(d_z));
  free(h_x);
  free(h_y);
  free(h_u);
  free(h_z);
  #endif
 
  return 0;
}